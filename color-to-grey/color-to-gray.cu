#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;

__global__ void color_to_greyscale(unsigned char* grayImage,
                                   unsigned char* rgbImage, int width,
                                   int height) {
    const int channels = 3;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * channels;
        unsigned char r = rgbImage[rgbOffset];
        unsigned char g = rgbImage[rgbOffset + 1];
        unsigned char b = rgbImage[rgbOffset + 2];
        grayImage[grayOffset] = 0.2126f * r + 0.7152f * g + 0.0722f * b;
    }
}

unsigned char* CopyArrayToGPU(unsigned char* HostArray, int NumElements) {
    int bytes = sizeof(unsigned char) * NumElements;
    unsigned char* DeviceArray;

    // Allocate memory on the GPU for array
    if (hipMalloc(&DeviceArray, bytes) != hipSuccess) {
        std::cout << "CopyArrayToGPU(): Couldn't allocate mem for array on GPU." << std::endl;
        return NULL;
    }

    // Copy the contents of the host array to the GPU
    if (hipMemcpy(DeviceArray, HostArray, bytes, hipMemcpyHostToDevice) !=
        hipSuccess) {
        std::cout << "CopyArrayToGPU(): Couldn't copy host array to GPU." << std::endl;
        hipFree(DeviceArray);
        return NULL;
    }

    return DeviceArray;
}

int main(int argc, char** argv) {
    Mat image;
    image = imread(argv[1], cv::IMREAD_COLOR);  // Read the file

    cv::cvtColor(image, image, cv::COLOR_BGR2RGB);

    if (!image.data)  // Check for invalid input
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    namedWindow("Display window",
                WINDOW_AUTOSIZE);     // Create a window for display.
    imshow("Display window", image);  // Show our image inside it.

    waitKey(0);

    const int width = image.cols;
    const int height = image.rows;
    unsigned char* color_array = image.ptr<unsigned char>();
    unsigned char* d_color_array = CopyArrayToGPU(color_array, image.elemSize());
    std::cout << "width " << width << ", height " << height << ", size " << image.size();
    unsigned char* grey_array = image.ptr<unsigned char>();
    unsigned char* d_grey_array = CopyArrayToGPU(grey_array, width * height);
    color_to_greyscale<<<1, 1>>>(d_grey_array, d_color_array, width, height);

    hipDeviceSynchronize();
    unsigned char* conv_grey_array = image.ptr<unsigned char>();
    hipMemcpy(conv_grey_array, d_grey_array, height * width * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    cv::Mat grey_image(height, width, CV_8UC1);
    std::memcpy(grey_image.data, conv_grey_array,
                height * width * sizeof(unsigned char));

    imshow("Display window", grey_image);

    waitKey(0);

    hipFree(d_color_array);
    hipFree(d_grey_array);

    return 0;
}
